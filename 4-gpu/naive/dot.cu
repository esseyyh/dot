#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAX_THREADS 1024
#define NUM_REPETITIONS 5

__global__ void matvec_kernel(float *A, float *B, float *C, int M, int K) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < M) {
    float sum = 0.0f;
    for (int k = 0; k < K; ++k) {
      sum += A[row * K + k] * B[k];
    }
    C[row] = sum;
  }
}

void matvec(float *A, float *B, float *C, int M, int K) {
  float *d_A, *d_B, *d_C;

  size_t size_A = M * K * sizeof(float);
  size_t size_B = K * sizeof(float);
  size_t size_C = M * sizeof(float);

  hipMalloc(&d_A, size_A);
  hipMalloc(&d_B, size_B);
  hipMalloc(&d_C, size_C);

  hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

  int block_size = 256;
  int grid_size = (M + block_size - 1) / block_size;

  matvec_kernel<<<grid_size, block_size>>>(d_A, d_B, d_C, M, K);

  hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

double get_time() {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(int argc, char *argv[]) {
  int sizes[][2] = {{128, 128},   {512, 512},   {1024, 1024},  {2048, 2048},
                    {4096, 4096}, {8192, 8192}, {16384, 16384}};
  int num_sizes = sizeof(sizes) / sizeof(sizes[0]);
  int num_threads = 4; // Default value
  if (argc > 1) {
    num_threads = atoi(argv[1]);
    if (num_threads <= 0 || num_threads > MAX_THREADS) {
      fprintf(stderr, "Invalid number of threads. Using default (4).\n");
      num_threads = 4;
    }
  }
  srand(time(NULL));

  printf("m,k,time,gflops\n");

  double best_gflops = 0.0;
  int best_m = 0, best_k = 0;

  for (int i = 0; i < num_sizes; i++) {
    int M = sizes[i][0];
    int K = sizes[i][1];
    float *A = (float *)malloc(M * K * sizeof(float));
    float *B = (float *)malloc(K * sizeof(float));
    float *C = (float *)malloc(M * sizeof(float));
    if (!A || !B || !C) {
      fprintf(stderr, "Memory allocation failed\n");
      exit(1);
    }

    for (int j = 0; j < M * K; j++) {
      A[j] = (float)rand() / RAND_MAX;
    }
    for (int j = 0; j < K; j++) {
      B[j] = (float)rand() / RAND_MAX;
    }

    double total_time = 0.0;
    double min_time = DBL_MAX;

    for (int rep = 0; rep < NUM_REPETITIONS; rep++) {
      double start_time = get_time();
      matvec(A, B, C, M, K);
      double end_time = get_time();
      double elapsed_time = end_time - start_time;

      total_time += elapsed_time;
      if (elapsed_time < min_time) {
        min_time = elapsed_time;
      }
    }

    double avg_time = total_time / NUM_REPETITIONS;
    double flops = 2.0 * M * K;
    double avg_gflops = flops / (avg_time * 1e9);
    double max_gflops = flops / (min_time * 1e9);

    printf("%d,%d,%.6f,%.2f\n", M, K, avg_time, avg_gflops);

    if (max_gflops > best_gflops) {
      best_gflops = max_gflops;
      best_m = M;
      best_k = K;
    }

    free(A);
    free(B);
    free(C);
  }

  printf("\nBest configuration:\n");
  printf("M=%d, K=%d\n", best_m, best_k);
  printf("Best performance: %.2f GFLOPS\n", best_gflops);

  return 0;
}
